#include "hip/hip_runtime.h"
#include "Matrix.cuh"
#include <cstring>
#include <fstream>
#define Zero ZeroCPU
#define PRINT_LOG false
#define TARGET_RESIDUE 0.000001

Matrix::Matrix(int cols, int rows) : cols(cols), rows(rows)
{
	if (PRINT_LOG) printf("Matrix constructor\n");
	hipMallocManaged(&mat, cols * rows * sizeof(double));
}

unsigned Matrix::getRows() const
{
	return rows;
}

unsigned Matrix::getCols() const
{
	return cols;
}

Matrix::Matrix(int cols, int rows, double* mat) : cols(cols), rows(rows), mat(mat)
{
	if (PRINT_LOG) printf("Matrix constructor\n");
	//hipMallocManaged(&mat, cols * rows * sizeof(double));
}

Matrix::Matrix(const Matrix& a)
{
	if (PRINT_LOG) printf("Matrix copy constructor\n");
	rows = a.rows;
	cols = a.cols;
	hipMallocManaged(&mat, cols * rows * sizeof(double));
	std::memcpy(mat, a.mat, cols * rows * sizeof(double));
}

void Matrix::operator=(const Matrix& a)
{
	if (PRINT_LOG) printf("Matrix assignment operator\n");
	rows = a.rows;
	cols = a.cols;
	hipFree(mat);
	hipMallocManaged(&mat, cols * rows * sizeof(double));
	std::memcpy(mat, a.mat, cols * rows * sizeof(double));
}

Matrix Matrix::Stub()
{
	return Matrix(1, 1);
}

Matrix Matrix::ZeroCPU(int cols, int rows)
{
	double* mat;
	hipMallocManaged(&mat, cols * rows * sizeof(double));
	for (long i = 0; i < cols * rows; i++)
	{
		mat[i] = 0.0f;
	}
	return Matrix(cols, rows, mat);
}

Matrix Matrix::OneCPU(int cols, int rows)
{
	double* mat;
	hipMallocManaged(&mat, cols * rows * sizeof(double));
	for (long i = 0; i < cols * rows; i++)
	{
		mat[i] = 1.0f;
	}
	return Matrix(cols, rows, mat);
}

/*__global__ void ZeroGPUKernel(const int n, double* A)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
	{
		A[index] = 0.0f;
	}
}

Matrix Matrix::ZeroGPU(int cols, int rows)
{
	Matrix* ret = new Matrix(cols, rows);
	int blockCount = (cols * rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
	ZeroGPUKernel <<<blockCount, BLOCK_SIZE >>>(cols * rows, ret->mat);
	hipDeviceSynchronize();
	return ret;
}*/

Matrix Matrix::IdentityCPU(int cols, int rows)
{
	if (cols != rows) throw "Identity matrix must be square";
	auto ret = Zero(cols, rows);
	for (int i = 0; i < cols; ++i)
	{
		ret.mat[i * cols + i] = 1.0f;
	}
	return ret;
}

Matrix Matrix::FromFile(std::string path)
{
	std::fstream reader;
	int cols, rows;
	reader.open(path, std::ios::in);
	reader.seekp(0);
	reader >> cols;
	reader >> rows;
	double* mat;
	hipMallocManaged(&mat, cols * rows * sizeof(double));
	for (int i = 0; i < cols * rows; ++i)
	{
		reader >> mat[i];
	}
	reader.close();
	return Matrix(cols, rows, mat);
}

Matrix Matrix::Jacobi(const Matrix& A, const Matrix& b)
{
	auto LU = A;
	auto invD = (LU.separateDiagonal());
	auto x = ZeroCPU(1, A.getRows());
	invD.inverseDiagonalInPlaceCPU();
	auto M = -invD * LU;
	auto temp = invD * b;
	double res;
	int counter = 9;

	do
	{
		x = (M * x + temp);
		if (counter++ == 9)
		{
			counter = 0;
			res = (A * x - b).vectorEuclideanNorm();
		}
	}
	while (res > TARGET_RESIDUE);
	return x;
}

Matrix Matrix::ForwardSubstitution(const Matrix& A, const Matrix& b)
{
	if (!(A.cols == A.rows && A.rows == b.rows)) throw "Incorrect dimensions";
	auto x = Matrix(1, A.getRows());

	for (int i = 0; i < x.rows; ++i)
	{
		double sum = 0;
		for (int j = 0; j < i; ++j)
		{
			sum += A.mat[i * A.cols + j] * x.mat[j];
		}
		x.mat[i] = (b.mat[i] - sum) / A.mat[i * A.cols + i];
	}
	return x;
}

Matrix Matrix::BackwardSubstitution(const Matrix& A, const Matrix& b)
{
	if (!(A.cols == A.rows && A.rows == b.rows)) throw "Incorrect dimensions";
	auto x = Matrix(1, A.getRows());

	x.mat[0] = b.mat[0] / A.mat[0];

	for (int i = x.rows - 1; i >= 0; --i)
	{
		double sum = 0;
		for (int j = i+1; j < A.cols; ++j)
		{
			sum += A.mat[i * A.cols + j] * x.mat[j];
		}
		x.mat[i] = (b.mat[i] - sum) / A.mat[i * A.cols + i];
	}
	return x;
}

Matrix Matrix::GaussSeidel(const Matrix& A, const Matrix& b)
{
	auto DL = -(A.lowerCPU() + A.diagonalCPU());
	auto U = A.upperCPU();
	auto x = ZeroCPU(1, A.getRows());
	auto temp = Matrix::ForwardSubstitution(DL, b);
	double res;
	int counter = 9;

	do
	{
		//x = -(Matrix::ForwardSubstitution(DL, U * x)) + temp;
		x = (Matrix::ForwardSubstitution(DL, U * x)) + temp;
		//if (counter++ == 9)
		//{
		//	counter = 0;
		res = (A * (-x) - b).vectorEuclideanNorm();
		//}
		//printf("res: %f \n", res);
		//(x).print();
	}
	while (res > TARGET_RESIDUE);
	return -x;
}

Matrix Matrix::LUMehtod(const Matrix& A, const Matrix& b)
{
	Matrix L = Matrix::Stub();
	Matrix U = Matrix::Stub();

	Matrix::doolitle(L, U, A);

	auto y = Matrix::ForwardSubstitution(L, b);

	return Matrix::BackwardSubstitution(U, y);
}

void Matrix::doolitle(Matrix& L, Matrix& U, const Matrix& A)
{
	if (A.cols != A.rows) throw "Matrix is not square";
	L = OneCPU(A.cols, A.rows).diagonalCPU();
	U = ZeroCPU(A.cols, A.rows);
	for (int j = 0; j < A.cols; ++j)
	{
		for (int i = 0; i <= j; ++i)
		{
			double sum = 0;
			for (int k = 0; k < i; ++k)
			{
				sum += L.mat[i * L.cols + k] * U.mat[k * U.cols + j];
			}
			U.mat[i * U.cols + j] = A.mat[i * U.cols + j] - sum;
		}

		for (int i = j + 1; i < A.cols; ++i)
		{
			double sum = 0;
			for (int k = 0; k < j; ++k)
			{
				sum += L.mat[i * L.cols + k] * U.mat[k * U.cols + j];
			}
			L.mat[i * U.cols + j] = 1 / U.mat[j * U.cols + j] * (A.mat[i * U.cols + j] - sum);
		}
	}
}

void Matrix::toFile(std::string path)
{
	std::fstream writer;
	writer.open(path, std::ios::out);
	writer.seekg(0);
	writer << cols << ' ' << rows << '\n';
	for (int i = 0; i < rows; ++i)
	{
		for (int j = 0; j < cols; ++j)
		{
			writer << mat[i * cols + j] << ' ';
		}
		writer << "\n";
	}
	writer.close();
}

Matrix Matrix::separateDiagonal()
{
	if (cols != rows) throw "Matrix is not square";
	auto ret = Zero(cols, rows);
	for (int i = 0; i < cols; ++i)
	{
		ret.mat[i * cols + i] = mat[i * cols + i];
		mat[i * cols + i] = 0.0f;
	}
	return ret;
}

Matrix Matrix::diagonalCPU() const
{
	if (cols != rows) throw "Matrix is not square";
	auto ret = Zero(cols, rows);
	for (int i = 0; i < cols; ++i)
	{
		ret.mat[i * cols + i] = mat[i * cols + i];
	}
	return ret;
}

Matrix Matrix::lowerCPU() const
{
	if (cols != rows) throw "Matrix is not square";
	auto ret = Zero(cols, rows);
	for (int j = 0; j < cols; ++j)
	{
		for (int i = 0; i < j; ++i)
		{
			ret.mat[j * cols + i] = mat[j * cols + i];
		}
	}
	return ret;
}

Matrix Matrix::upperCPU() const
{
	if (cols != rows) throw "Matrix is not square";
	auto ret = Zero(cols, rows);
	for (int j = 0; j < cols; ++j)
	{
		for (int i = j + 1; i < cols; ++i)
		{
			ret.mat[j * cols + i] = mat[j * cols + i];
		}
	}
	return ret;
}

void Matrix::inverseDiagonalInPlaceCPU()
{
	if (cols != rows) throw "Matrix is not square";
	for (int i = 0; i < cols; ++i)
	{
		if (mat[i * cols + i] == 0) throw "0 on diagonal";
		mat[i * cols + i] = 1 / mat[i * cols + i];
	}
}

void Matrix::transposeVectorInPlace()
{
	unsigned int tmp = cols;
	cols = rows;
	rows = tmp;
}

double Matrix::vectorEuclideanNorm()
{
	if (cols != 1 && rows != 1) throw "Matrix is not a vector";
	double sum = 0;
	for (int i = 0; i < cols * rows; ++i)
	{
		sum += mat[i] * mat[i];
	}
	return sqrt(sum);
}

Matrix Matrix::lu()
{
	throw "Not implemented";
}

void Matrix::print() const
{
	for (int i = 0; i < rows; ++i)
	{
		for (int j = 0; j < cols; ++j)
		{
			printf("%f ", mat[i * cols + j]);
		}
		printf("\n");
	}
	printf("\n");
}

Matrix::~Matrix()
{
	if (PRINT_LOG) printf("Matrix destructor\n");
	hipFree(mat);
	//free(mat);
}

__global__ void mulKernel(const int commonDim, const int cols, const int n, double* A, double* B, double* C)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int j = index; j < n; j += stride)
	{
		int row = j / cols;
		int col = j % cols;
		C[j] = 0;
		for (int i = 0; i < commonDim; i++)
		{
			C[j] += A[row * commonDim + i] * B[i * cols + col];
		}
	}
}

Matrix operator*(const Matrix& a, const Matrix& b)
{
	if (a.cols != b.rows) throw "wrong dimensions for multiplication";
	double* mat;
	hipMallocManaged(&mat, b.cols * a.rows * sizeof(double));
	int blockCount = (a.rows * b.cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
	if (PRINT_LOG) printf("Matrix multiplication on %d blocks x %d threads\n", blockCount, BLOCK_SIZE);
	mulKernel <<< blockCount, BLOCK_SIZE >>>(a.cols, b.cols, b.cols * a.rows, a.mat, b.mat, mat);
	hipDeviceSynchronize();
	return Matrix(b.cols, a.rows, mat);
}

__global__ void addKernel(const int n, double* A, double* B, double* C)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int j = index; j < n; j += stride)
	{
		C[j] = A[j] + B[j];
	}
}

Matrix operator+(const Matrix& a, const Matrix& b)
{
	if (a.cols != b.cols || a.rows != b.rows) throw "dimensions must equal for addition";
	double* mat;
	hipMallocManaged(&mat, a.cols * a.rows * sizeof(double));
	int blockCount = (a.cols * a.rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
	if (PRINT_LOG) printf("Matrix addition on %d blocks x %d threads\n", blockCount, BLOCK_SIZE);
	addKernel <<< blockCount, BLOCK_SIZE >>>(a.cols * a.rows, a.mat, b.mat, mat);
	hipDeviceSynchronize();
	return Matrix(a.cols, a.rows, mat);
}

__global__ void subKernel(const int n, double* A, double* B, double* C)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int j = index; j < n; j += stride)
	{
		C[j] = A[j] - B[j];
	}
}

Matrix operator-(const Matrix& a, const Matrix& b)
{
	if (a.cols != b.cols || a.rows != b.rows) throw "dimensions must equal for addition";
	double* mat;
	hipMallocManaged(&mat, a.cols * a.rows * sizeof(double));
	int blockCount = (a.cols * a.rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
	if (PRINT_LOG) printf("Matrix addition on %d blocks x %d threads\n", blockCount, BLOCK_SIZE);
	subKernel <<< blockCount, BLOCK_SIZE >>>(a.cols * a.rows, a.mat, b.mat, mat);
	hipDeviceSynchronize();
	return Matrix(a.cols, a.rows, mat);
}

__global__ void additiveInverseKernel(const int n, double* A, double* B)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int j = index; j < n; j += stride)
	{
		A[j] = -B[j];
	}
}

Matrix operator-(const Matrix& a)
{
	double* mat;
	hipMallocManaged(&mat, a.cols * a.rows * sizeof(double));
	int blockCount = (a.cols * a.rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
	additiveInverseKernel <<<blockCount, BLOCK_SIZE >>>(a.cols * a.rows, mat, a.mat);
	hipDeviceSynchronize();
	return Matrix(a.cols, a.rows, mat);
}
