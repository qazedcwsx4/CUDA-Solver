#include "hip/hip_runtime.h"
#include "Matrix.cuh"

#define Zero ZeroCPU

Matrix::Matrix(int cols, int rows) : cols(cols), rows(rows)
{
	hipMallocManaged(&mat, cols * rows * sizeof(float));
}

Matrix* Matrix::ZeroCPU(int cols, int rows)
///zdaje mi sie, ze metoda bedzie wykonywania stosunkowo niewiele razy wiec nie potrzebuje zrownoleglenia.
{
	Matrix* ret = new Matrix(cols, rows);

	for (long i = 0; i < cols * rows; i++)
	{
		ret->mat[i] = 0.0f;
	}

	return ret;
}

Matrix* Matrix::OneCPU(int cols, int rows)
///zdaje mi sie, ze metoda bedzie wykonywania stosunkowo niewiele razy wiec nie potrzebuje zrownoleglenia.
{
	Matrix* ret = new Matrix(cols, rows);

	for (long i = 0; i < cols * rows; i++)
	{
		ret->mat[i] = 1.0f;
	}

	return ret;
}

__global__ void ZeroGPUKernel(const int n, float* A)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
	{
		A[index] = 0.0f;
	}
}

Matrix* Matrix::ZeroGPU(int cols, int rows)
{
	Matrix* ret = new Matrix(cols, rows);
	int blockCount = (cols * rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
	ZeroGPUKernel <<<blockCount, BLOCK_SIZE >>>(cols * rows, ret->mat);
	hipDeviceSynchronize();
	return ret;
}

__global__ void mulKernel(const int commonDim, const int cols, float* A, float* B, float* C)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	int row = index / cols;
	int col = index % cols;

	C[index] = 0;
	for (int i = 0; i < commonDim; i++)
	{
		C[index] += A[row * commonDim + i] * B[i * cols + col];
	}
}

Matrix* Matrix::operator*(const Matrix* b) const
{
	if (this->cols != b->rows) throw "wrong dimensions for multiplication";
	auto ret = new Matrix(this->rows, b->cols);
	int blockCount = (this->rows * b->cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
	mulKernel <<< blockCount, BLOCK_SIZE >>>(this->cols, ret->cols, this->mat, b->mat, ret->mat);
	hipDeviceSynchronize();
	return ret;
}


Matrix* Matrix::separateDiagonal()
{
	if (cols != rows) throw "Matrix is not square";
	Matrix* ret = Matrix::ZeroCPU(cols, rows);
	for (int i = 0; i < cols; ++i)
	{
		ret->mat[i * cols + i] = this->mat[i * cols + i];
		this->mat[i * cols + i] = 0.0f;
	}
	return ret;
}

Matrix* Matrix::lu()
{
}

void Matrix::print() const
{
	for (int i = 0; i < rows; ++i)
	{
		for (int j = 0; j < cols; ++j)
		{
			printf("%f ", mat[i * cols + j]);
		}
		printf("\n");
	}
}

Matrix::~Matrix()
{
	hipFree(mat);
}

Matrix* operator*(const Matrix& a, const Matrix* b)
{
	return nullptr;
}
