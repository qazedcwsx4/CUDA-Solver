#include "hip/hip_runtime.h"
#include ""
#include <chrono>

#include <stdio.h>
#include "Matrix.cuh"
#include <vector>


int main()
{
	Matrix A = Matrix::Stub();
	Matrix b = Matrix::Stub();
	Matrix x = Matrix::Stub();

	//Matrix::createTask(A, b, 994);
	//Matrix::createTask(A, b, 3000);
	Matrix::createTest(A, b, x, 1000);


	//wywolanie przed zeby przygotowac device
	Matrix::JacobiOptimal(A, b);
	auto start = std::chrono::steady_clock::now();
	x = Matrix::JacobiOptimal(A, b);
	auto end = std::chrono::steady_clock::now();
	printf("Jacobi method: %lld us\n", std::chrono::duration_cast<std::chrono::microseconds>(end - start).count());

	/*Matrix::GaussSeidelOptimal(A, b);
	start = std::chrono::steady_clock::now();
	x = Matrix::GaussSeidelOptimal(A, b);
	end = std::chrono::steady_clock::now();
	printf("Gauss-Seidel method: %lld us\n", std::chrono::duration_cast<std::chrono::microseconds>(end - start).count());

	Matrix::LUMehtodOptimal(A, b);
	start = std::chrono::steady_clock::now();
	x = Matrix::LUMehtodOptimal(A, b);
	end = std::chrono::steady_clock::now();
	printf("LU method: %lld us\n", std::chrono::duration_cast<std::chrono::microseconds>(end - start).count());
	printf("residue: %f\n",(A * x - b).vectorEuclideanNorm());*/



	//system("pause");
	return 0;
}
